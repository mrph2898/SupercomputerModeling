#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "mpi.h"
#include <string.h>

#include "hip/hip_runtime_api.h"
#define A1 0.0
#define A2 4.0
#define B1 0.0
#define B2 3.0

#define epsilon 1e-5
#define threadsPerBlock 4
#define numThreadsX 2
#define numThreadsY 2
#define EPS_REL  1e-6
#define DOWN_TAG 1000
#define MAX_ITER 100000

__device__ double dev_u_2(double x, double y){
    return sqrt(4 + x * y);
}

__device__ double dev_k_3(double x, double y){
    return 4 + x + y;
}

__device__ double dev_q_2(double x, double y){
    double sum = x + y;
    if (sum < 0) {
        return 0;
    } else {
        return sum;
    }
}

__device__ double dev_F(double x, double y){
    return ((pow(x, 3) - x*x*(y - 4) - x*(y*y + 8) +
             y*(y*y + 4*y - 8) + 4*dev_q_2(x, y)*pow((4 + x*y), 2)) /
            (4 * pow((4 + x*y), 1.5)));
}

__device__ double dev_psi_R(double x, double y){
    return (y*(4 + x + y) + 2*(4 + x*y)) / (2*sqrt(4 + x*y));
}

__device__ double dev_psi_L(double x, double y){
    return (-y*(4 + x + y) + 2*(4 + x*y)) / (2*sqrt(4 + x*y));
}


__device__ double dev_psi_T(double x, double y){
    return (x*(4 + x + y)) / (2*sqrt(4 + x*y));
}


__device__ double dev_psi_B(double x, double y){
    return -dev_psi_T(x, y);
}

__global__ void init_w(int n_y, double **w){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    w[tid_x][tid_y] = 0.0;
    return;
}

__global__ void copy_interior_w(int M, int N,
                                double **w,
                                double **w_pr){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (tid_x == 0 || tid_y == 0 || tid_x == M + 1 || tid_y == N + 1) {
        w_pr[tid_x][tid_y] = 0.0;
    } else {
        w_pr[tid_x][tid_y] = w[tid_x][tid_y];
    }
    return;
}

__global__ void get_top(int n_x, int n_y,
                        int x_idx, int y_idx,
                        double **w,
                        double *dev_t_send){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    if (tid_y == (y_idx + n_y - 1))
        dev_t_send[i] = w[i+1][n_y];
}

__global__ void get_bottom(int n_x, int n_y,
                           int x_idx, int y_idx,
                           double **w,
                           double *dev_b_send){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    if (tid_y == y_idx)
        dev_b_send[i] = w[i+1][1];
}
__global__ void get_left(int n_x, int n_y,
                         int x_idx, int y_idx,
                         double **w,
                         double *dev_l_send){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int j = tid_y - y_idx;
    if (tid_x == x_idx)
        dev_l_send[j] = w[1][j+1];
}
__global__ void get_right(int n_x, int n_y,
                          int x_idx, int y_idx,
                          double **w,
                          double *dev_r_send){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int j = tid_y - y_idx;
    if (tid_x == (x_idx + n_x - 1))
        dev_r_send[j] = w[n_x][j+1];
}

__global__ void set_top(int n_x, int n_y,
                        int x_idx, int y_idx,
                        double **w,
                        double *dev_t_recv)
{
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    if (tid_y == (y_idx + n_y - 1))
        w[i][n_y + 1] = dev_t_recv[i - 1];
}

__global__ void set_bottom(int n_x, int n_y,
                           int x_idx, int y_idx,
                           double **w,
                           double *dev_b_recv
){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    if (tid_y == y_idx)
        w[i][0] = dev_b_recv[i-1];
}
__global__ void set_left(int n_x, int n_y,
                         int x_idx, int y_idx,
                         double **w,
                         double *dev_l_recv
){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int j = tid_y - y_idx;
    if (tid_x == x_idx)
        w[0][j] = dev_l_recv[j-1];
}
__global__ void set_right(int n_x, int n_y,
                          int x_idx, int y_idx,
                          double **w,
                          double *dev_r_recv
){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int j = tid_y - y_idx;
    if (tid_x == (x_idx + n_x - 1))
        w[n_x+1][j] = dev_r_recv[j - 1];
}


__global__ void preset_top(int n_x, int n_y,
                           int x_idx, int y_idx,
                           double **w,
                           double h1, double h2
){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    if (tid_y == (y_idx + n_y - 1))
        w[i][n_y + 1] = dev_u_2(A1 + (x_idx + i - 1)*h1,
                                B1 + (y_idx + n_y) * h2);
}

__global__ void preset_bottom(int n_x, int n_y,
                              int x_idx, int y_idx,
                              double **w,
                              double h1, double h2
){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    if (tid_y == y_idx)
        w[i][0] = dev_u_2(A1 + (x_idx + i - 1)*h1,
                          B1 + (y_idx - 1) * h2);
}
__global__ void preset_left(int n_x, int n_y,
                            int x_idx, int y_idx,
                            double **w,
                            double h1, double h2
){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int j = tid_y - y_idx;
    if (tid_x == x_idx)
        w[0][j] = dev_u_2(A1 + (x_idx - 1)*h1,
                          B1 + (y_idx + j - 1) * h2);
}
__global__ void preset_right(int n_x, int n_y,
                             int x_idx, int y_idx,
                             double **w,
                             double h1, double h2
){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int j = tid_y - y_idx;
    if (tid_x == (x_idx + n_x - 1))
        w[n_x+1][j] = dev_u_2(A1 + (x_idx + n_x)*h1,
                              B1 + (y_idx + j - 1) * h2);
}


__global__ void cudaB_right(int M, int N, double **B,
                            int x_idx, int y_idx,
                            double h1, double h2,
                            double x_start, double y_start,
                            int left_border, int right_border,
                            int top_border, int bottom_border){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    int j = tid_y - y_idx;

    B[i][j] = dev_F(x_start + (i - 1) * h1, y_start + (j - 1) * h2);

    if (left_border){
        B[1][j] = (dev_F(x_start, y_start + (j - 1) * h2) +
                   dev_psi_L(x_start, y_start + (j - 1) * h2) * 2/h1);
    } else if (right_border){
        B[M][j] = (dev_F(x_start + (M - 1)*h1, y_start + (j - 1) * h2) +
                   dev_psi_R(x_start + (M - 1)*h1, y_start + (j - 1) * h2) * 2/h1);
    }
    if (top_border){
        B[i][N] = (dev_F(x_start + (i - 1)*h1, y_start + (N - 1)*h2) +
                   dev_psi_T(x_start + (i - 1)*h1, y_start + (N - 1)*h2) * 2/h2);
    } else if (bottom_border){
        B[i][1] = (dev_F(x_start + (i - 1)*h1, y_start) +
                   dev_psi_B(x_start + (i - 1)*h1, y_start) * 2/h2);
    }
    if (left_border && top_border){
        B[1][N] = (dev_F(x_start, y_start + (N - 1)*h2) +
                   (2/h1 + 2/h2) * (dev_psi_L(x_start, y_start + (N - 1)*h2) +
                                    dev_psi_T(x_start, y_start + (N - 1)*h2)) / 2);
    } else if (left_border && bottom_border){
        B[1][1] =  (dev_F(x_start, y_start)
                    + (2/h1 + 2/h2) * (dev_psi_L(x_start, y_start) + dev_psi_B(x_start, y_start)) / 2);
    } else if (right_border && top_border){
        B[M][N] = (dev_F(x_start + (M - 1)*h1, y_start + (N - 1)*h2) +
                   (2/h1 + 2/h2) * (dev_psi_R(x_start + (M - 1)*h1, y_start + (N - 1)*h2) +
                                    dev_psi_T(x_start + (M - 1)*h1, y_start + (N - 1)*h2)) / 2);
    } else if (right_border && bottom_border){
        B[M][1] = (dev_F(x_start + (M - 1)*h1, y_start) +
                   (2/h1 + 2/h2) * (dev_psi_R(x_start + (M - 1)*h1, y_start) +
                                    dev_psi_B(x_start + (M - 1)*h1, y_start)) / 2);
    }
}

__device__ double dev_aw_x_ij(int N,
                              double **w,
                              double x_start, double y_start,
                              int i, int j,
                              double h1, double h2
){
    return (1/h1) * (dev_k_3(x_start + (i + 0.5 - 1) * h1,y_start + (j - 1) * h2) * (w[i + 1][j] - w[i][j]) / h1
                     - dev_k_3(x_start + (i - 0.5 - 1) * h1,y_start + (j - 1) * h2) * (w[i][j] - w[i - 1][j]) / h1);
}

__device__ double dev_aw_ij(int N,
                            double **w,
                            double x_start, double y_start,
                            int i, int j,
                            double h1, double h2
){
    return (dev_k_3(x_start + (i - 0.5 - 1) * h1,y_start + (j - 1) * h2) * (w[i][j] - w[i - 1][j]) / h1);
}

__device__ double dev_bw_y_ij(int N,
                              double **w,
                              double x_start, double y_start,
                              int i, int j,
                              double h1, double h2
){
    return (1/h2) * (dev_k_3(x_start + (i - 1) * h1,y_start + (j + 0.5 - 1) * h2) * (w[i][j + 1] - w[i][j]) / h2
                     - dev_k_3(x_start + (i - 1) * h1,y_start + (j - 0.5 - 1) * h2) * (w[i][j] - w[i][j - 1]) / h2);
}

__device__ double dev_bw_ij(int N,
                            double **w,
                            double x_start, double y_start,
                            int i, int j,
                            double h1, double h2
){
    return (dev_k_3(x_start + (i - 1) * h1,y_start + (j - 0.5 - 1) * h2) * (w[i][j] - w[i][j-1]) / h2);
}

__global__ void cuda_Aw_mult(int M, int N,
                             int x_idx, int y_idx,
                             double **A, double **w,
                             double h1, double h2,
                             double x_start, double y_start,
                             int left_border, int right_border,
                             int top_border, int bottom_border
) {
    double aw_x, bw_y;
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    int j = tid_y - y_idx;

    if (( i == 0) || i == M+1 || j == 0 || j == N+1){
        A[i][j] = w[i][j];
    } else {
        aw_x = dev_aw_x_ij(N, w, x_start, y_start, i, j, h1, h2);
        bw_y = dev_bw_y_ij(N, w, x_start, y_start, i, j, h1, h2);
        A[i][j] = -aw_x - bw_y + dev_q_2(x_start + (i - 1) * h1,
                                         y_start + (j - 1) * h2) * w[i][j];
    }

    // Left interior border filling
    if (left_border){
        aw_x = dev_aw_ij(N, w, x_start, y_start, 2, j , h1, h2);
        bw_y = dev_bw_y_ij(N, w, x_start, y_start, 1, j, h1, h2);
        A[1][j] = -2*aw_x / h1 - bw_y + (dev_q_2(x_start, y_start + (j - 1) * h2)
                                         + 2/h1) * w[1][j];
    } else if (right_border){
        // Right interior border
        aw_x = dev_aw_ij(N, w, x_start, y_start, M, j, h1, h2);
        bw_y = dev_bw_y_ij(N, w, x_start, y_start, M, j, h1, h2);
        A[M][j] = 2*aw_x / h1 - bw_y + (dev_q_2(x_start + (M - 1) * h1,
                                                y_start + (j - 1) * h2) + 2/h1) * w[M][j];
    }

    // Top border
    if (top_border){
        aw_x = dev_aw_x_ij(N, w, x_start, y_start, i, N, h1, h2);
        bw_y = dev_bw_ij(N, w, x_start, y_start, i, N, h1, h2);
        A[i][N] = -aw_x + 2*bw_y / h2 + dev_q_2(x_start + (i - 1) * h1,
                                                y_start + (N - 1) * h2) * w[i][N];
    } else if (bottom_border){
        // Bottom border
        aw_x = dev_aw_x_ij(N, w, x_start, y_start, i, 1, h1, h2);
        bw_y = dev_bw_ij(N, w, x_start, y_start, i, 2, h1, h2);
        A[i][1] = -aw_x - 2*bw_y / h2 + dev_q_2(x_start + (i - 1)* h1, y_start) * w[i][1];
    }
    if (left_border && bottom_border){
        aw_x = dev_aw_ij(N, w, x_start, y_start, 2, 1, h1, h2);
        bw_y = dev_bw_ij(N, w, x_start, y_start, 1, 2, h1, h2);
        A[1][1] = -2*aw_x / h1 - 2*bw_y / h2 + (dev_q_2(x_start, y_start) + 2/h1) * w[1][1];
    } else if (left_border && top_border){
        aw_x = dev_aw_ij(N, w, x_start, y_start, 2, N, h1, h2);
        bw_y = dev_bw_ij(N, w, x_start, y_start, 1, N, h1, h2);
        A[1][N] = -2*aw_x / h1 + 2*bw_y / h2 + (dev_q_2(x_start, y_start + (N - 1) * h2) + 2/h1)* w[1][N];
    }
    if (right_border && bottom_border){
        aw_x = dev_aw_ij(N, w, x_start, y_start, M, 1, h1, h2);
        bw_y = dev_bw_ij(N, w, x_start, y_start, M, 2, h1, h2);
        A[M][1] = 2*aw_x / h1 - 2 * bw_y / h2 + (dev_q_2(x_start + (M - 1) * h1, y_start) + 2/h1) * w[M][1];
    } else if (right_border && top_border) {
        aw_x = dev_aw_ij(N, w, x_start, y_start, M, N, h1, h2);
        bw_y = dev_bw_ij(N, w, x_start, y_start, M, N, h1, h2);
        A[M][N] = 2*aw_x / h1 + 2 * bw_y / h2 + (dev_q_2(x_start + (M - 1) * h1,
                                                         y_start + (N - 1) * h2) + 2/h1) * w[M][N];
    }
}

__global__ void calculate_r(int M, int N,
                            int x_idx, int y_idx,
                            double **r,
                            double **Aw,
                            double **B){
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    int j = tid_y - y_idx;

    if(i == 0 || i == M+1 || j == 0 || j == N+1)
        r[i][j] = 0;
    else
        r[i][j] = Aw[i][j] - B[i][j];
}


__device__ double dev_rho_1(int i,
                            int M,
                            int left_border,
                            int right_border){
    if ((left_border && i == 1) || (right_border && i == M))
        return 0.5;
    return 1;
}

__device__ double dev_rho_2(int j,
                            int N,
                            int bottom_border,
                            int top_border){
    if ((bottom_border && j == 1) || (top_border && j == N))
        return   0.5;
    return 1;
}

__global__ void cuda_dot_product(int n_x, int n_y,
                                 int x_idx, int y_idx,
                                 double **U, double **V,
                                 double h1, double h2,
                                 int left_border, int right_border,
                                 int top_border, int bottom_border,
                                 double *partial_product
){
//    int num_threads_x = (int) sqrt(threadsPerBlock);
//    int num_threads_y = threadsPerBlock / numThreadsX;
    __shared__ double cache[numThreadsX];
//    __shared__ double cache_y[numThreadsY];


    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    int j = tid_y - y_idx;

    int cacheIndex_x = threadIdx.x;
//    int cacheIndex_y = threadIdx.y;
    double temp = 0;
    double rho, r1, r2;

    while (i < n_x) {
        j = tid_y - y_idx;
        while (j < n_y) {
            r1 = dev_rho_1(i, n_x, left_border, right_border);
            r2 = dev_rho_2(j, n_y, bottom_border, top_border);
            rho = r1 * r2;
            double part_dot = (rho * U[i][j] * V[i][j] * h1 * h2);
            temp += part_dot;
            j += blockDim.y * gridDim.y;
        }
        i += blockDim.x * gridDim.x;
    }
    cache[cacheIndex_x] = temp;
    __syncthreads();
    int k = blockDim.x / 2;
    while (k > 0) {
        if (cacheIndex_x < k) {
            cache[cacheIndex_x] += cache[cacheIndex_x + k];
        }
        __syncthreads();
        k = k / 2;
    }
    if (cacheIndex_x == 0) {
        partial_product[blockIdx.x] = cache[0];
    }
    return;
}

__global__ void cuda_w_step(int n_y,
                            int x_idx, int y_idx,
                            double **w,
                            double **r_k,
                            double tau
//                            double *w_next
) {
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int i = tid_x - x_idx;
    int j = tid_y - y_idx;
    double r_k_scaled = r_k[i][j] * tau;
    w[i][j] = w[i][j] - r_k_scaled;
    return;
}


void get_idx_n_idx(int *idx,
                   int *n_idx,
                   int process_amnt,
                   int grid_size,
                   int coordinate){
    if (grid_size % process_amnt == 0) {
        *n_idx = grid_size / process_amnt;
        *idx = coordinate * (grid_size / process_amnt);
    }
    else
    {
        if (coordinate == 0){
            *n_idx = grid_size % process_amnt + grid_size / process_amnt;
            *idx = 0;
        } else
        {
            *n_idx = grid_size / process_amnt;
            *idx = grid_size % process_amnt + coordinate * (grid_size / process_amnt);
        }
    }
}


void send_recv_borders(int n_x, int n_y,
                       const int process_amounts[2],
                       double x_idx,
                       double y_idx,
                       const int my_coords[2],
                       int tag,
                       double **w,
                       double *b_send,
                       double *l_send,
                       double *t_send,
                       double *r_send,
                       double *b_rec,
                       double *l_rec,
                       double *t_rec,
                       double *r_rec,
                       int left_border, int right_border,
                       int top_border, int bottom_border,
                       double h1, double h2,
                       MPI_Comm MPI_COMM_CART
){
    int neighbour_coords[2];
    int neighbour_rank;
//    int num_threads_x = (int) sqrt(threadsPerBlock);
//    int num_threads_y = threadsPerBlock / numThreadsX;
    int blocksPerGrid_x = n_x / numThreadsX + 1;
    int blocksPerGrid_y = n_y / numThreadsY + 1;
    dim3 gridShape = dim3(blocksPerGrid_x, blocksPerGrid_y);
    dim3 blockShape = dim3(numThreadsX, numThreadsY);

    MPI_Request request[4] = {MPI_REQUEST_NULL, MPI_REQUEST_NULL,
                              MPI_REQUEST_NULL, MPI_REQUEST_NULL};
    MPI_Status status;

    double *dev_b_send, *dev_l_send, *dev_t_send, *dev_r_send;
    double *dev_b_rec, *dev_l_rec, *dev_t_rec, *dev_r_rec;
    hipMalloc((void**)&dev_b_send, sizeof(double[n_x]));
    hipMalloc((void**)&dev_t_send, sizeof(double[n_x]));
    hipMalloc((void**)&dev_b_rec, sizeof(double[n_x]));
    hipMalloc((void**)&dev_t_rec, sizeof(double[n_x]));

    hipMalloc((void**)&dev_l_send, sizeof(double[n_y]));
    hipMalloc((void**)&dev_r_send, sizeof(double[n_y]));
    hipMalloc((void**)&dev_l_rec, sizeof(double[n_y]));
    hipMalloc((void**)&dev_r_rec, sizeof(double[n_y]));
    /////////////////
    // Bottom border send
    if ((process_amounts[1] > 1) && !bottom_border) {
        get_bottom<<<gridShape, blockShape>>>(n_x, n_y,
                                           x_idx, y_idx,
                                           w, dev_b_send);
        hipMemcpy(b_send, dev_b_send,
                   sizeof(double[n_x]), hipMemcpyDeviceToHost);

        neighbour_coords[0] = my_coords[0];
        neighbour_coords[1] = my_coords[1] - 1;

        MPI_Cart_rank(MPI_COMM_CART, neighbour_coords, &neighbour_rank);
        MPI_Isend(b_send, n_x, MPI_DOUBLE,
                  neighbour_rank, tag + DOWN_TAG,
                  MPI_COMM_CART, &request[0]);
    }

    // Left border send
    if ((process_amounts[0] > 1) && !left_border) {
        get_left<<<gridShape, blockShape>>>(n_x, n_y,
                                              x_idx, y_idx,
                                              w, dev_l_send);
        hipMemcpy(l_send, dev_l_send,
                   sizeof(double[n_y]), hipMemcpyDeviceToHost);

        neighbour_coords[0] = my_coords[0] - 1;
        neighbour_coords[1] = my_coords[1];

        MPI_Cart_rank(MPI_COMM_CART, neighbour_coords, &neighbour_rank);
        MPI_Isend(l_send, n_y, MPI_DOUBLE,
                  neighbour_rank, tag,
                  MPI_COMM_CART, &request[1]);
    }

    // Top border
    if ((process_amounts[1] > 1) && !top_border) {
        get_top<<<gridShape, blockShape>>>(n_x, n_y,
                                            x_idx, y_idx,
                                            w, dev_t_send);
        hipMemcpy(t_send, dev_t_send,
                   sizeof(double[n_x]), hipMemcpyDeviceToHost);

        neighbour_coords[0] = my_coords[0];
        neighbour_coords[1] = my_coords[1] + 1;

        MPI_Cart_rank(MPI_COMM_CART, neighbour_coords, &neighbour_rank);
        MPI_Isend(t_send, n_x, MPI_DOUBLE,
                  neighbour_rank, tag,
                  MPI_COMM_CART, &request[2]);
    }

    // Right border
    if ((process_amounts[0] > 1) && !right_border) {
        get_right<<<gridShape, blockShape>>>(n_x, n_y,
                                            x_idx, y_idx,
                                            w, dev_r_send);
        hipMemcpy(r_send, dev_r_send,
                   sizeof(double[n_y]), hipMemcpyDeviceToHost);

        neighbour_coords[0] = my_coords[0] + 1;
        neighbour_coords[1] = my_coords[1];

        MPI_Cart_rank(MPI_COMM_CART, neighbour_coords, &neighbour_rank);
        MPI_Isend(r_send, n_y, MPI_DOUBLE,
                  neighbour_rank, tag,
                  MPI_COMM_CART, &request[3]);
    }

    // Receive borders
    // Bottom border
    if ((bottom_border && (process_amounts[1] > 1)) || (process_amounts[1] == 1)) {
        preset_bottom<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx, w, h1, h2);
    } else {
        neighbour_coords[0] = my_coords[0];
        neighbour_coords[1] = my_coords[1] - 1;
        MPI_Cart_rank(MPI_COMM_CART, neighbour_coords, &neighbour_rank);
        MPI_Recv(b_rec, n_x, MPI_DOUBLE,
                 neighbour_rank, tag, MPI_COMM_CART, &status);

        hipMemcpy(dev_b_rec, b_rec,
                   sizeof(double[n_x]), hipMemcpyHostToDevice);
        set_bottom<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx, w, dev_b_rec);
    }

    // Left border
    if ((left_border && (process_amounts[0] > 1)) || (process_amounts[0] == 1)) {
        preset_left<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx, w, h1, h2);

    } else {
        neighbour_coords[0] = my_coords[0] - 1;
        neighbour_coords[1] = my_coords[1];

        MPI_Cart_rank(MPI_COMM_CART, neighbour_coords, &neighbour_rank);
        MPI_Recv(l_rec, n_y, MPI_DOUBLE,
                 neighbour_rank, tag, MPI_COMM_CART, &status);

        hipMemcpy(dev_l_rec, l_rec,
                   sizeof(double[n_y]), hipMemcpyHostToDevice);
        set_left<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx, w, dev_l_rec);
    }

    // Top border
    if ((top_border && (process_amounts[1] > 1)) || (process_amounts[1] == 1)) {
        preset_top<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx, w, h1, h2);
    } else {
        neighbour_coords[0] = my_coords[0];
        neighbour_coords[1] = my_coords[1] + 1;
        MPI_Cart_rank(MPI_COMM_CART, neighbour_coords, &neighbour_rank);
        MPI_Recv(t_rec, n_x, MPI_DOUBLE,
                 neighbour_rank, tag + DOWN_TAG,
                 MPI_COMM_CART, &status);

        hipMemcpy(dev_t_rec, t_rec,
                   sizeof(double[n_x]), hipMemcpyHostToDevice);
        set_top<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx, w, dev_t_rec);
    }

    // Right border
    if ((right_border && (process_amounts[0] > 1)) || (process_amounts[0] == 1)) {
        preset_right<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx, w, h1, h2);
    } else {
        neighbour_coords[0] = my_coords[0] + 1;
        neighbour_coords[1] = my_coords[1];
        MPI_Cart_rank(MPI_COMM_CART, neighbour_coords, &neighbour_rank);
        MPI_Recv(r_rec, n_y, MPI_DOUBLE,
                 neighbour_rank, tag, MPI_COMM_CART, &status);

        hipMemcpy(dev_r_rec, r_rec,
                   sizeof(double[n_y]), hipMemcpyHostToDevice);
        set_right<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx, w, dev_r_rec);
    }

    for (int i = 0; i < 4; i++) {
        MPI_Wait(&request[i], &status);
    }
}


void cudaDotProduct(int n_x, int n_y,
                    int x_idx, int y_idx,
                    double **U, double **V,
                    double h1, double h2,
                    int left_border, int right_border,
                    int top_border, int bottom_border,
                    double *curr_sum)
{
//    int num_threads_x = (int) sqrt(threadsPerBlock);
//    int num_threads_y = threadsPerBlock / numThreadsX;
    int blocksPerGrid_x = n_x / numThreadsX + 1;
    int blocksPerGrid_y = n_y / numThreadsY + 1;
    dim3 gridShape = dim3(blocksPerGrid_x, blocksPerGrid_y);
    dim3 blockShape = dim3(numThreadsX, numThreadsY);
    /////////
    double c, *partial_c;
    double *dev_partial_c;
    partial_c = (double*) calloc(blocksPerGrid_x, sizeof(double));
    // Allocate device memory
    hipMalloc((void**)&dev_partial_c, blocksPerGrid_x * sizeof(double));
    /////////
    cuda_dot_product<<<gridShape, blockShape>>>(n_x, n_y, x_idx, y_idx,
                                                U, V, h1, h2,
                                                left_border, right_border,
                                                top_border, bottom_border,
                                                dev_partial_c);
    hipMemcpy(partial_c, dev_partial_c,
               blocksPerGrid_x * sizeof(double),
               hipMemcpyDeviceToHost);
    c = 0;
    for (int i = 0; i < blocksPerGrid_x; ++i) {
        c = c + partial_c[i];
    }
    (*curr_sum) = c;
    /////////
    hipFree(dev_partial_c);
    free(partial_c);
    return;
}


int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Program receive %d numbers. Should be 2: M, N\n", argc);
        return -1;
    }

    int M = atoi(argv[argc - 2]);
    int N = atoi(argv[argc - 1]);
    if ((M <= 0) || (N <= 0)) {
        printf("M and N should be integer and > 0!!!\n");
        return -1;
    }
    printf("M = %d, N = %d\n", M, N);
    int my_rank, n_processes;
    int process_amounts[2] = {0, 0};
    int write[1] = {0};
    double h1 = (A2 - A1) / M;
    double h2 = (B2 - B1) / N;
    double cur_eps = 1.0;

    MPI_Init(&argc, &argv);
    MPI_Status status;

    // For the cartesian topology
    MPI_Comm MPI_COMM_CART;
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &n_processes);

    // Creating rectangular supports
    MPI_Dims_create(n_processes, 2, process_amounts);
    int periods[2] = {0, 0};

    // Create cartesian  topology in communicator
    MPI_Cart_create(MPI_COMM_WORLD, 2,
                    process_amounts, periods,
                    1, &MPI_COMM_CART);

    int my_coords[2];
    // Receive corresponding to rank process coordinates
    MPI_Cart_coords(MPI_COMM_CART, my_rank, 2, my_coords);

    int x_idx, n_x;
    get_idx_n_idx(&x_idx, &n_x, process_amounts[0], M+1, my_coords[0]);

    int y_idx, n_y;
    get_idx_n_idx(&y_idx, &n_y, process_amounts[1], N+1, my_coords[1]);

    double start_time = MPI_Wtime();
    ////////////////////////
    hipProfilerStart();
//    int num_threads_x = (int) sqrt(threadsPerBlock);
//    int num_threads_y = threadsPerBlock / numThreadsX;
    int blocksPerGrid_x = n_x / numThreadsX + 1;
    int blocksPerGrid_y = n_y / numThreadsY + 1;
    dim3 gridShape = dim3(blocksPerGrid_x, blocksPerGrid_y);
    dim3 blockShape = dim3(numThreadsX, numThreadsY);
    ////////////////////////
    double *t_send = (double *) malloc(sizeof(double[n_x]));
    double *t_rec = (double *) malloc(sizeof(double[n_x]));
    double *b_send = (double *) malloc(sizeof(double[n_x]));
    double *b_rec = (double *) malloc(sizeof(double[n_x]));

    double *l_send = (double *) malloc(sizeof(double[n_y]));
    double *l_rec = (double *) malloc(sizeof(double[n_y]));
    double *r_send = (double *) malloc(sizeof(double[n_y]));
    double *r_rec = (double *) malloc(sizeof(double[n_y]));
    int n_iters = 0;
    double block_eps;

    double **w, **w_pr, **B;
    double **Aw, **r_k, **Ar, **w_w_pr;

    hipMalloc((void**)&w, sizeof(double[n_x + 2][n_y + 2]));
    hipMalloc((void**)&w_pr, sizeof(double[n_x + 2][n_y + 2]));
    hipMalloc((void**)&B, sizeof(double[n_x + 2][n_y + 2]));
    hipMalloc((void**)&Aw, sizeof(double[n_x + 2][n_y + 2]));
    hipMalloc((void**)&r_k, sizeof(double[n_x + 2][n_y + 2]));
    hipMalloc((void**)&Ar, sizeof(double[n_x + 2][n_y + 2]));
    hipMalloc((void**)&w_w_pr, sizeof(double[n_x + 2][n_y + 2]));
    ////////////////////////
    double tau = 0;
    double global_tau = 0;
    double denumenator;
    double whole_denum;
//    double global_alpha, global_beta;
//    double eps_local, eps_r;
    int left_border = 0;
    int top_border = 0;
    int right_border = 0;
    int bottom_border = 0;
    if (my_coords[0] == 0)
        left_border = 1;

    if (my_coords[0] == (process_amounts[0] - 1))
        right_border = 1;

    if (my_coords[1] == 0)
        bottom_border = 1;

    if (my_coords[1] == (process_amounts[1] - 1))
        top_border = 1;
    ////////////////////////
    cudaB_right<<<gridShape, blockShape>>>(n_x, n_y, B,
                                           x_idx, y_idx,
                                           h1, h2,
                                           A1 + x_idx * h1,
                                           B1 + y_idx * h2,
                                           left_border, right_border,
                                           top_border,  bottom_border);
    init_w<<<gridShape, blockShape>>>(n_x, w);

    int tag = 0;
    while ((cur_eps > EPS_REL) && (n_iters < MAX_ITER)) {
        if (my_rank == 0) {
            if (n_iters % 1000 == 0)
                printf("%g \n", cur_eps);
        }
        n_iters++;

        copy_interior_w<<<gridShape, blockShape>>>(n_x, n_y,
                                                   w, w_pr);

        send_recv_borders(n_x, n_y, process_amounts,
                          x_idx, y_idx, my_coords, tag,
                          w,
                          b_send, l_send, t_send, r_send,
                          b_rec, l_rec, t_rec, r_rec,
                          left_border, right_border,
                          top_border, bottom_border,
                          h1, h2, MPI_COMM_CART);
        cuda_Aw_mult<<<gridShape, blockShape>>>(n_x, n_y,
                    x_idx, y_idx,
                    Aw, w,
                    h1, h2,
                    A1 + x_idx * h1, B1 + y_idx * h2,
                    left_border, right_border,
                    top_border,  bottom_border);

        calculate_r<<<gridShape, blockShape>>>(n_x, n_y,
                                               x_idx, y_idx,
                                               r_k, Aw, B);
        send_recv_borders(n_x, n_y, process_amounts,
                          x_idx, y_idx, my_coords, tag,
                          r_k,
                          b_send, l_send, t_send, r_send,
                          b_rec, l_rec, t_rec, r_rec,
                          left_border, right_border,
                          top_border, bottom_border,
                          h1, h2, MPI_COMM_CART);
        cuda_Aw_mult<<<gridShape, blockShape>>>(n_x, n_y,
                     x_idx, y_idx,
                     Ar, r_k,
                     h1, h2,
                     A1 + x_idx * h1, B1 + y_idx * h2,
                     left_border, right_border,
                     top_border,  bottom_border);
        cudaDotProduct(n_x, n_y,
                     x_idx, y_idx,
                     Ar, r_k, h1, h2,
                     left_border, right_border,
                     top_border, bottom_border,
                     &tau);

        cudaDotProduct(n_x, n_y,
                       x_idx, y_idx,
                       Ar, Ar, h1, h2,
                       left_border, right_border,
                       top_border, bottom_border,
                       &denumenator);
        MPI_Allreduce(&tau,  &global_tau, 1,
                      MPI_DOUBLE, MPI_SUM, MPI_COMM_CART);
        MPI_Allreduce(&denumenator,  &whole_denum, 1,
                      MPI_DOUBLE, MPI_SUM, MPI_COMM_CART);
        global_tau = global_tau / whole_denum;
        cuda_w_step<<<gridShape, blockShape>>>(n_y,
                    x_idx, y_idx,
                    w, r_k,
                    tau
//                    w_next
                    );
        calculate_r<<<gridShape, blockShape>>>(n_x, n_y,
                                               x_idx, y_idx,
                                               w_w_pr, w, w_pr);

        cudaDotProduct(n_x, n_y,
                       x_idx, y_idx,
                       w_w_pr, w_w_pr, h1, h2,
                       left_border, right_border,
                       top_border, bottom_border,
                       &block_eps);
        block_eps = sqrt(block_eps);

        MPI_Allreduce(&block_eps, &cur_eps, 1,
                      MPI_DOUBLE, MPI_SUM, MPI_COMM_CART);
    }

    // Waiting for all processes
    MPI_Barrier(MPI_COMM_WORLD);
    double end_time = MPI_Wtime();

    if (my_rank != 0) {
        MPI_Recv(write, 1, MPI_INT, my_rank - 1, 0, MPI_COMM_WORLD, &status);
    } else {
        printf("TIME = %f\n", end_time - start_time);
        printf("Number of iterations = %d\n", n_iters);
        printf("Tau = %f\n", tau);
        printf("Eps = %f\n", EPS_REL);
    }

    if (my_rank != n_processes - 1)
        MPI_Send(write, 1, MPI_INT, my_rank + 1, 0, MPI_COMM_WORLD);


    hipFree(w);
    hipFree(w_pr);
    hipFree(B);
    hipFree(Aw);
    hipFree(r_k);
    hipFree(Ar);
    hipFree(w_w_pr);

    free(t_send);
    free(t_rec);
    free(b_send);
    free(b_rec);
    free(r_send);
    free(r_rec);
    free(l_send);
    free(l_rec);
    hipProfilerStop();
    MPI_Finalize();
    return 0;
}